#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010 http://vladx.net/
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "config.h"


__constant__ uint target[4];
__constant__ uint k[64];
__constant__ uint rconst[16];


static const uint k_cpu[64] =
{
	0xd76aa478,	0xe8c7b756,	0x242070db,	0xc1bdceee,
	0xf57c0faf,	0x4787c62a,	0xa8304613,	0xfd469501,
	0x698098d8,	0x8b44f7af,	0xffff5bb1,	0x895cd7be,
	0x6b901122,	0xfd987193,	0xa679438e,	0x49b40821,

	0xf61e2562,	0xc040b340,	0x265e5a51,	0xe9b6c7aa,
	0xd62f105d,	0x2441453,	0xd8a1e681,	0xe7d3fbc8,
	0x21e1cde6,	0xc33707d6,	0xf4d50d87,	0x455a14ed,
	0xa9e3e905,	0xfcefa3f8,	0x676f02d9,	0x8d2a4c8a,

	0xfffa3942,	0x8771f681,	0x6d9d6122,	0xfde5380c,
	0xa4beea44,	0x4bdecfa9,	0xf6bb4b60,	0xbebfbc70,
	0x289b7ec6,	0xeaa127fa,	0xd4ef3085,	0x4881d05,
	0xd9d4d039,	0xe6db99e5,	0x1fa27cf8,	0xc4ac5665,

	0xf4292244,	0x432aff97,	0xab9423a7,	0xfc93a039,
	0x655b59c3,	0x8f0ccc92,	0xffeff47d,	0x85845dd1,
	0x6fa87e4f,	0xfe2ce6e0,	0xa3014314,	0x4e0811a1,
	0xf7537e82,	0xbd3af235,	0x2ad7d2bb,	0xeb86d391,
};

static const uint rconst_cpu[16] = {7, 12, 17, 22, 5, 9, 14, 20, 4, 11, 16, 23, 6, 10, 15, 21};


__device__ inline uint leftrotate (uint x, uint c)
{
	return (x << c) | (x >> (32-c));
}

__device__ inline void step (const uint i, const uint f, const uint g, uint &a, uint &b, uint &c, uint &d, const uint * w)
{
	uint temp = d;
	d = c;
	c = b;
	b = b + leftrotate((a + f + k[i] + w[g]), rconst[(i / 16) * 4 + i % 4]);
	a = temp;
}

__device__ inline void md5 (const uint * w, uint &a, uint &b, uint &c, uint &d)
{
	const uint a0 = 0x67452301;
	const uint b0 = 0xEFCDAB89;
	const uint c0 = 0x98BADCFE;
	const uint d0 = 0x10325476;
	
	a = a0;
	b = b0;
	c = c0;
	d = d0;
	
	uint f, g;
	uint i = 0;
	
	for(; i != 16; i++)
	{
		f = (b & c) | ((~b) & d);
		g = i;
		step(i, f, g, a, b, c, d, w);
	}
	
	/*
	f = (b & c) | ((~b) & d);
	g = 0;
	step(0, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 1;
	step(1, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 2;
	step(2, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 3;
	step(3, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 4;
	step(4, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 5;
	step(5, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 6;
	step(6, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 7;
	step(7, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 8;
	step(8, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 9;
	step(9, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 10;
	step(10, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 11;
	step(11, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 12;
	step(12, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 13;
	step(13, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 14;
	step(14, f, g, a, b, c, d, w);
	
	f = (b & c) | ((~b) & d);
	g = 15;
	step(15, f, g, a, b, c, d, w);
	*/
	
	for(; i != 32; i++)
	{
		f = (d & b) | ((~d) & c);
		g = (5*i + 1) % 16;
		step(i, f, g, a, b, c, d, w);
	}
	
	/*
	f = (d & b) | ((~d) & c);
	g = 1;
	step(16, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 6;
	step(17, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 11;
	step(18, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 0;
	step(19, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 5;
	step(20, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 10;
	step(21, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 15;
	step(22, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 4;
	step(23, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 9;
	step(24, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 14;
	step(25, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 3;
	step(26, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 8;
	step(27, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 13;
	step(28, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 2;
	step(29, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 7;
	step(30, f, g, a, b, c, d, w);
	
	f = (d & b) | ((~d) & c);
	g = 12;
	step(31, f, g, a, b, c, d, w);
	*/
	
	for(; i != 48; i++)
	{
		f = b ^ c ^ d;
		g = (3*i + 5) % 16;
		step(i, f, g, a, b, c, d, w);
	}
	
	/*
	f = b ^ c ^ d;
	g = 5;
	step(32, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 8;
	step(33, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 11;
	step(34, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 14;
	step(35, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 1;
	step(36, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 4;
	step(37, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 7;
	step(38, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 10;
	step(39, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 13;
	step(40, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 0;
	step(41, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 3;
	step(42, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 6;
	step(43, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 9;
	step(44, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 12;
	step(45, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 15;
	step(46, f, g, a, b, c, d, w);
	
	f = b ^ c ^ d;
	g = 2;
	step(47, f, g, a, b, c, d, w);
	*/
	
	for(; i != 64; i++)
	{
		f = c ^ (b | (~d));
		g = (7*i) % 16;
		step(i, f, g, a, b, c, d, w);
	}
	
	/*
	f = c ^ (b | (~d));
	g = 0;
	step(48, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 7;
	step(49, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 14;
	step(50, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 5;
	step(51, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 12;
	step(52, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 3;
	step(53, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 10;
	step(54, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 1;
	step(55, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 8;
	step(56, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 15;
	step(57, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 6;
	step(58, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 13;
	step(59, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 4;
	step(60, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 11;
	step(61, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 2;
	step(62, f, g, a, b, c, d, w);
	
	f = c ^ (b | (~d));
	g = 9;
	step(63, f, g, a, b, c, d, w);
	*/
	
	a += a0;
	b += b0;
	c += c0;
	d += d0;
}

__device__ inline void mdset_f (uint &a, uint &b, uint &c, uint &d, const uint x, const uint s, const uint ac)
{
	a += (((b) & (c)) | ((~b) & (d))) + x + ac;
	a = (a << s) | (a >> (32 - s));
	a += b;
}

__device__ inline void mdset_g (uint &a, uint &b, uint &c, uint &d, const uint x, const uint s, const uint ac)
{
	a += (((b) & (d)) | ((c) & (~d))) + x + ac;
	a = (a << s) | (a >> (32 - s));
	a += b;
}

__device__ inline void mdset_h (uint &a, uint &b, uint &c, uint &d, const uint x, const uint s, const uint ac)
{
	a += ((b) ^ (c) ^ (d)) + x + ac;
	a = (a << s) | (a >> (32 - s));
	a += b;
}

__device__ inline void mdset_i (uint &a, uint &b, uint &c, uint &d, const uint x, const uint s, const uint ac)
{
	a += ((c) ^ ((b) | (~d))) + x + ac;
	a = (a << s) | (a >> (32 - s));
	a += b;
}

__device__ inline void md5v2 (const uint * in, uint &a, uint &b, uint &c, uint &d)
{
	const uint a0 = 0x67452301;
	const uint b0 = 0xEFCDAB89;
	const uint c0 = 0x98BADCFE;
	const uint d0 = 0x10325476;
	
	a = a0;
	b = b0;
	c = c0;
	d = d0;
	
	mdset_f(a, b, c, d, in[0],  rconst[0], k[0]);
	mdset_f(d, a, b, c, in[1],  rconst[1], k[1]);
	mdset_f(c, d, a, b, in[2],  rconst[2], k[2]);
	mdset_f(b, c, d, a, in[3],  rconst[3], k[3]);
	mdset_f(a, b, c, d, in[4],  rconst[0], k[4]);
	mdset_f(d, a, b, c, in[5],  rconst[1], k[5]);
	mdset_f(c, d, a, b, in[6],  rconst[2], k[6]);
	mdset_f(b, c, d, a, in[7],  rconst[3], k[7]);
	mdset_f(a, b, c, d, in[8],  rconst[0], k[8]);
	mdset_f(d, a, b, c, in[9],  rconst[1], k[9]);
	mdset_f(c, d, a, b, in[10], rconst[2], k[10]);
	mdset_f(b, c, d, a, in[11], rconst[3], k[11]);
	mdset_f(a, b, c, d, in[12], rconst[0], k[12]);
	mdset_f(d, a, b, c, in[13], rconst[1], k[13]);
	mdset_f(c, d, a, b, in[14], rconst[2], k[14]);
	mdset_f(b, c, d, a, in[15], rconst[3], k[15]);
	
	mdset_g(a, b, c, d, in[1],  rconst[4], k[16]);
	mdset_g(d, a, b, c, in[6],  rconst[5], k[17]);
	mdset_g(c, d, a, b, in[11], rconst[6], k[18]);
	mdset_g(b, c, d, a, in[0],  rconst[7], k[19]);
	mdset_g(a, b, c, d, in[5],  rconst[4], k[20]);
	mdset_g(d, a, b, c, in[10], rconst[5], k[21]);
	mdset_g(c, d, a, b, in[15], rconst[6], k[22]);
	mdset_g(b, c, d, a, in[4],  rconst[7], k[23]);
	mdset_g(a, b, c, d, in[9],  rconst[4], k[24]);
	mdset_g(d, a, b, c, in[14], rconst[5], k[25]);
	mdset_g(c, d, a, b, in[3],  rconst[6], k[26]);
	mdset_g(b, c, d, a, in[8],  rconst[7], k[27]);
	mdset_g(a, b, c, d, in[13], rconst[4], k[28]);
	mdset_g(d, a, b, c, in[2],  rconst[5], k[29]);
	mdset_g(c, d, a, b, in[7],  rconst[6], k[30]);
	mdset_g(b, c, d, a, in[12], rconst[7], k[31]);
	
	mdset_h(a, b, c, d, in[5],  rconst[8], k[32]);
	mdset_h(d, a, b, c, in[8],  rconst[9], k[33]);
	mdset_h(c, d, a, b, in[11], rconst[10], k[34]);
	mdset_h(b, c, d, a, in[14], rconst[11], k[35]);
	mdset_h(a, b, c, d, in[1],  rconst[8], k[36]);
	mdset_h(d, a, b, c, in[4],  rconst[9], k[37]);
	mdset_h(c, d, a, b, in[7],  rconst[10], k[38]);
	mdset_h(b, c, d, a, in[10], rconst[11], k[39]);
	mdset_h(a, b, c, d, in[13], rconst[8], k[40]);
	mdset_h(d, a, b, c, in[0],  rconst[9], k[41]);
	mdset_h(c, d, a, b, in[3],  rconst[10], k[42]);
	mdset_h(b, c, d, a, in[6],  rconst[11], k[43]);
	mdset_h(a, b, c, d, in[9],  rconst[8], k[44]);
	mdset_h(d, a, b, c, in[12], rconst[9], k[45]);
	mdset_h(c, d, a, b, in[15], rconst[10], k[46]);
	mdset_h(b, c, d, a, in[2],  rconst[11], k[47]);
	
	mdset_i(a, b, c, d, in[0],  rconst[12], k[48]);
	mdset_i(d, a, b, c, in[7],  rconst[13], k[49]);
	mdset_i(c, d, a, b, in[14], rconst[14], k[50]);
	mdset_i(b, c, d, a, in[5],  rconst[15], k[51]);
	mdset_i(a, b, c, d, in[12], rconst[12], k[52]);
	mdset_i(d, a, b, c, in[3],  rconst[13], k[53]);
	mdset_i(c, d, a, b, in[10], rconst[14], k[54]);
	mdset_i(b, c, d, a, in[1],  rconst[15], k[55]);
	mdset_i(a, b, c, d, in[8],  rconst[12], k[56]);
	mdset_i(d, a, b, c, in[15], rconst[13], k[57]);
	mdset_i(c, d, a, b, in[6],  rconst[14], k[58]);
	mdset_i(b, c, d, a, in[13], rconst[15], k[59]);
	mdset_i(a, b, c, d, in[4],  rconst[12], k[60]);
	mdset_i(d, a, b, c, in[11], rconst[13], k[61]);
	mdset_i(c, d, a, b, in[2],  rconst[14],  k[62]);
	mdset_i(b, c, d, a, in[9],  rconst[15], k[63]);
	
	a += a0;
	b += b0;
	c += c0;
	d += d0;
}

__global__ void _gpu_md5_bruteforce (uint * words, int * res)
{
	uint idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint a, b, c, d, in[16];
	
	/*
	for (i = 0; i < 16; i++)
		in[i] = words[idx * 16 + i];
	*/
	
	/* Разворачиваем цикл, описанный выше */
	
	const uint tmp = idx * 16;
	
	in[0] = words[tmp];
	in[1] = words[tmp + 1];
	in[2] = words[tmp + 2];
	in[3] = words[tmp + 3];
	in[4] = words[tmp + 4];
	in[5] = words[tmp + 5];
	in[6] = words[tmp + 6];
	in[7] = words[tmp + 7];
	in[8] = words[tmp + 8];
	in[9] = words[tmp + 9];
	in[10] = words[tmp + 10];
	in[11] = words[tmp + 11];
	in[12] = words[tmp + 12];
	in[13] = words[tmp + 13];
	in[14] = words[tmp + 14];
	in[15] = words[tmp + 15];
	
	md5(in, a, b, c, d);
	
	if (target[0] == a && target[1] == b && target[2] == c && target[3] == d)
		(* res) = idx;
}

extern "C"
{
void get_gpu_props (int * prop)
{
	hipDeviceProp_t deviceProp;
	int nDevCount, i;
	
	hipGetDeviceCount(&nDevCount);
	
	prop[0] = nDevCount;
	prop[1] = 0;
	prop[2] = 0;
	
	for (i = 0; i < nDevCount; i++)
	{
		if (hipSuccess != hipGetDeviceProperties(&deviceProp, i))
		{
			prop[1] += 64;
			prop[2] += 128;
			return;
		}
		prop[1] += deviceProp.multiProcessorCount;
		prop[2] += deviceProp.maxThreadsPerBlock;
	}
}

int * d_res;
uint * d_words;

void gpu_init (unsigned int * abcd, unsigned int dsize)
{
	hipMemcpyToSymbol(HIP_SYMBOL(target), abcd, sizeof(target));
	hipMemcpyToSymbol(HIP_SYMBOL(k), k_cpu, sizeof(k_cpu));
	hipMemcpyToSymbol(HIP_SYMBOL(rconst), rconst_cpu, sizeof(rconst_cpu));
	hipMalloc((void **) &d_res, sizeof(* d_res));
	hipMalloc((void **) &d_words, dsize);
}

int gpu_md5_bruteforce (uint * words, int * prop, unsigned int dsize)
{
	int blocks, threads_per_block, res;
	
	hipMemcpyAsync(d_words, words, dsize, hipMemcpyHostToDevice, 0);
	
	res = -1;
	hipMemcpyAsync(d_res, &res, sizeof(res), hipMemcpyHostToDevice, 0);
	
	blocks = prop[1];
	threads_per_block = prop[2];
	_gpu_md5_bruteforce <<<blocks, threads_per_block>>> (d_words, d_res);
	hipDeviceSynchronize();
	hipMemcpy(&res, d_res, sizeof(res), hipMemcpyDeviceToHost);
	
	return res;
}
}
