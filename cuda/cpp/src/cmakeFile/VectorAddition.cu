#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "VectorAddition.h"

// to prevent IDE complains about unknown CUDA keywords
#ifdef __CDT_PARSER__
#define __global__
#define __device__
#define __host__
#define __shared__
uint3 threadIdx;
uint3 blockIdx;
dim3 blockDim;
dim3 gridDim;
int warpSize;
#define CUDA_KERNEL_DIM(...)
#else
#define CUDA_KERNEL_DIM(...)  <<< __VA_ARGS__ >>>
#endif

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    if( err != hipSuccess )
    {
        std::cerr << "CUDA call failed at " << file << ":" << line << " : " << hipGetErrorString( err ) << std::endl;
        exit (EXIT_FAILURE);
    }
}

__global__ void cudaVecAdd( ScalarT* const C, const ScalarT* const A, const ScalarT* const B, size_t N )
{
    size_t gid = blockDim.x * blockIdx.x + threadIdx.x;
    if( gid < N )
        C[gid] = A[gid] + B[gid];
}

void vecAdd( ScalarT* const c, const ScalarT* const a, const ScalarT* const b, size_t N )
{
    hipError_t status;
    const size_t size = N * sizeof(ScalarT);
    // Prepare device memory //
    ScalarT* dev_A;
    ScalarT* dev_B;
    ScalarT* dev_C;

    status = hipMalloc( &dev_A, size );
    CudaSafeCall( status );

    status = hipMalloc( &dev_B, size );
    CudaSafeCall( status );

    status = hipMalloc( &dev_C, size );
    CudaSafeCall( status );

    status = hipMemcpy( dev_A, a, size, hipMemcpyHostToDevice );
    CudaSafeCall( status );

    status = hipMemcpy( dev_B, b, size, hipMemcpyHostToDevice );
    CudaSafeCall( status );

    // Call CUDA kernel //
    size_t threadsPerBlock = 32;
    size_t blocksPerGrid = ( N + threadsPerBlock - 1 ) / threadsPerBlock;
    std::cout << "Threads/Block: " << threadsPerBlock << std::endl;
    std::cout << "Blocks/Grid: " << blocksPerGrid << std::endl;

    cudaVecAdd CUDA_KERNEL_DIM( blocksPerGrid, threadsPerBlock )( dev_C, dev_A, dev_B, N );

    // Load result from device //
    status = hipMemcpy( c, dev_C, size, hipMemcpyDeviceToHost );
    CudaSafeCall( status );

    // Free CUDA memory //
    status = hipFree( dev_A );
    CudaSafeCall( status );

    status = hipFree( dev_B );
    CudaSafeCall( status );

    status = hipFree( dev_C );
    CudaSafeCall( status );
}
